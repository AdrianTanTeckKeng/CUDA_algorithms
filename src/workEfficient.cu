#include "hip/hip_runtime.h"
#include "cuda_device_runtime_api.h"
#include ""
#include "..\header\workEfficient.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>


__global__ void StreamCompaction::Common::kernMapToBoolean(int n, int* bools, const int* idata)
{
	int index = threadIdx.x + blockDim.x * blockIdx.x;
	if(index >= n)
		return;

	if (idata[index] != 0)
		bools[index] = 1;
	else
		bools[index] = 0;
}

__global__ void StreamCompaction::Common::kernScatter(int n, int* odata, const int* idata, const int* bools, const int* indices)
{
	int index = threadIdx.x + blockDim.x * blockIdx.x;
	if (index >= n)
		return;
	if (bools[index] != 0)
		odata[indices[index]] = idata[index];
}

namespace StreamCompaction
{
	namespace workEfficient
	{
		
		using StreamCompaction::Common::PerformanceTimer;
		PerformanceTimer& timer()
		{
			static PerformanceTimer timer;
			return timer;
		}
		

		__global__ void kernelUp(int size, int d, int* data)
		{
			int index = threadIdx.x + blockDim.x * blockIdx.x;
			if (index >= size)
				return;
			int stride = 1 << d;
			if ((index + 1) % (1 << (d+1)) == 0)
				data[index] += data[index - stride];
		}

		__global__ void kernelDown(int size, int d, int* data)
		{
			int index = threadIdx.x + blockDim.x * blockIdx.x;
			if(index >= size)
				return;
			int stride = 1 << d;
			if ((index + 1) % (1 << (d + 1)) == 0)
			{
				int temp = data[index - stride];
				data[index - stride] = data[index];
				data[index] += temp;
			}
				
		}

		void scan(int n, int* odata, const int* idata)
		{
			int* device;
			int size = 1 << ilog2ceil(n);
			hipMalloc((void**) &device, size * sizeof(int));
			hipMemcpy(device, idata, n * sizeof(int), hipMemcpyHostToDevice);
			hipMemset(device + n, 0, (size - n)*sizeof(int));

			dim3 BlocksPerGrid((size + blockSize - 1) / blockSize);

			// Start timer
			timer().startGpuTimer();

			// Now perform upsweep
			for (int d = 0; d <= ilog2ceil(n) - 1; d++)
			{
				kernelUp << <BlocksPerGrid, blockSize >> > (size, d, device);
			}
			// Zero out one element
			hipMemset(device + size - 1, 0, sizeof(int));

			// Then perform downsweep
			for (int d = ilog2ceil(n)-1; d >= 0; d--)
			{
				kernelDown << <BlocksPerGrid, blockSize >> > (size, d, device);
			}
			timer().endGpuTimer();

			hipMemcpy(odata, device, n * sizeof(int), hipMemcpyDeviceToHost);
			hipFree(device);
		}

		int compact(int n, int* odata, const int* idata)
		{
			int size = 1 << ilog2ceil(n);
			dim3 BlocksPerGrid((size + blockSize - 1) / blockSize);

			// Define all device pointers and allocate all memories on GPU
			int* d_input, * d_boolean, * d_indices, * d_output;
			
			// Allocate memory for boolean array
			hipMalloc((void**)&d_boolean, size * sizeof(int));
			hipMemset(d_boolean + n, 0, (size - n) * sizeof(int));

			// Allocate memory for input and transfer data from host to gpu
			hipMalloc((void**)&d_input, n * sizeof(int));
			hipMemcpy(d_input, idata, n * sizeof(int), hipMemcpyHostToDevice);

			// Allocate memory for output and indices
			hipMalloc((void**)&d_output, size * sizeof(int));
			hipMalloc((void**)&d_indices, size * sizeof(int));
			
			// Now start loop.
			// We first map input to boolean
			timer().startGpuTimer();
			StreamCompaction::Common::kernMapToBoolean<<<BlocksPerGrid, blockSize>>>(n, d_boolean ,d_input);

			// Now we perform scan on Boolean. We first map data from Boolean to indices(we will need the boolean still)
			hipMemcpy(d_indices, d_boolean, size * sizeof(int), hipMemcpyDeviceToDevice);
			
			// Then perform upsweep
			for (int d = 0; d <= ilog2ceil(n) - 1; d++)
			{
				kernelUp <<<BlocksPerGrid, blockSize >> > (size, d, d_indices);
			}
			// Zero out one element
			hipMemset(d_indices + size - 1, 0, sizeof(int));

			// Then perform downsweep
			for (int d = ilog2ceil(n) - 1; d >= 0; d--)
			{
				kernelDown << <BlocksPerGrid, blockSize >> > (size, d, d_indices);
			}
			

			// Now perform scatter
			StreamCompaction::Common::kernScatter << <BlocksPerGrid, blockSize >> > (n, d_output, d_input, d_boolean, d_indices);
			timer().endGpuTimer();
			
			// Retrieve all required data
			int* finalCount = new int[1];
			hipMemcpy(finalCount, d_indices + size - 1, sizeof(int), hipMemcpyDeviceToHost);
			int count = finalCount[0];
			hipMemcpy(odata, d_output, count * sizeof(int), hipMemcpyDeviceToHost);
			
			// Free data
			delete[] finalCount;
			
			hipFree(d_output);
			hipFree(d_input);
			hipFree(d_indices);
			hipFree(d_boolean);
			
			return count;
		}
	}
}
